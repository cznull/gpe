#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "hipfft/hipfft.h"

#include <stdio.h>
#include "gpe.h"

current2* d_u = nullptr, * d_uk = nullptr;
current* d_potantial = nullptr, * d_nr = nullptr, * d_power = nullptr;
current t = 0.0f;

hipfftHandle fftPlan;
hipfftResult fresu;

int cuinit(int size);

__global__ void evok2(current2* uk, int size, current dt, current halfinvereffm, current k2, current ns) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	current kx, ky, h, temp, c, s;
	kx = x > size / 2 ? x - size : x;
	ky = y > size / 2 ? y - size : y;
	h = (kx * kx + ky * ky) * k2 * halfinvereffm;;
	current2 u = uk[y * size + x];
	c = cos(h * dt);
	s = sin(h * dt);
//	c = rsqrt(1 + h * h * dt * dt);
//	s = c * h * dt;
	temp = u.x *c + u.y * s;
	u.y = (u.y * c - u.x * s) * ns;
	u.x = temp*ns;
	uk[y * size + x] = u;
}

//ns ����Ҷ�任��һ��
//k2 

__global__ void evov(current2* uk, int size, current* v, current* nr, current* power, current g, current r,current l,current dt,current t) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int index = y * size + x;
	current h, temp, c, s, ex, n, nrl, ledge;
	current2 u = uk[index];
	nrl = nr[index];
	n = u.x * u.x + u.y * u.y;
	h = v[index] + g * n + current(2.0) * g * nrl;
//	current mu = 0.74f * 40.0f * 40.0f;
//	current x0,y0;
//	x0 = 0.125f * cos(mu * t) * (1.0f - sin(mu * t));
//	y0 = 0.125f * cos(mu * t) * sin(mu * t);
//	h = h + 100.0f * 40.0f * 40.0f * exp(-(((x + 0.0f) / size - 0.5f - x0) * ((x + 0.0f) / size - 0.5f - x0) + ((y + 0.0f) / size - 0.5f - y0) * ((y + 0.0f) / size - 0.5f - y0)) * 17777.8f);
	c = cos(h * dt);
	s = sin(h * dt);
	ledge = max(0.0f, max(64.0f - x, 1.0f + 64.0f + x - size));
	ledge = max(ledge, max(64.0f - y, 1.0f + 64.0f + y - size));
	ex = current(0.5) * (r * nrl - l - l * ledge * 0.5);
	ex = exp(ex * dt);
//	ex = 1.0 + ex * dt;
	temp = u.x * c + u.y * s;
	u.y = u.y * c - u.x * s;
	u.y = u.y * ex;
	u.x = temp * ex;
	nrl = nrl + power[index]*dt;
	nrl = nrl * exp((-l - n * r) * dt);
	nr[index] = nrl;
	uk[index] = u;
}

int cuevo(int size, current dt, current g, current r, current l) {
	cufftExec(fftPlan, d_u, d_u, HIPFFT_FORWARD);
	evok2 << < dim3(size / 128, size, 1), dim3(128, 1, 1) >> > (d_u, size, dt, 0.5, 6.2832 * 6.2832, 1.0 / size / size);
	cufftExec(fftPlan, d_u, d_u, HIPFFT_BACKWARD);
	evov << < dim3(size / 128, size, 1), dim3(128, 1, 1) >> > (d_u, size, d_potantial, d_nr, d_power, g, r, l, dt, t);
	t += dt;
	return 0;
}

int getu(void* u, int size, int type) {
	switch (type) {
	case 0:
		hipMemcpy(u, d_u, size * size * sizeof(current2), hipMemcpyDeviceToHost);
		break;
	case 1:
		hipMemcpy(u, d_nr, size * size * sizeof(current), hipMemcpyDeviceToHost);
		break;
	default:
		break;
	}
	return 0;
}

int setu(void* u, int size) {
	hipMemcpy(d_u, u, size * size * sizeof(current2), hipMemcpyHostToDevice);
	return 0;
}

int setnr(void* nr, int size) {
	hipMemcpy(d_nr, nr, size * size * sizeof(current), hipMemcpyHostToDevice);
	return 0;
}

int setpotantial(void* u, int size) {
	hipMemcpy(d_potantial, u, size * size * sizeof(current), hipMemcpyHostToDevice);
	return 0;
}

int setpower(void* u, int size) {
	hipMemcpy(d_power, u, size * size * sizeof(current), hipMemcpyHostToDevice);
	return 0;
}


int cuinit(int size) {
	hipMalloc(&d_potantial, size * size * sizeof(current));
	hipMalloc(&d_nr, size * size * sizeof(current));
	hipMemset(d_nr, 0, size * size * sizeof(current));
	hipMalloc(&d_power, size * size * sizeof(current));
	hipMalloc(&d_u, size * size * sizeof(current2));
	hipMalloc(&d_uk, size * size * sizeof(current2));
	hipfftPlan2d(&fftPlan, size, size, CUFFT_cur2cur);
	return 0;
}